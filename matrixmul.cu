
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define VECTORSIZE 10

__global__ void matrixmul(int **A, int *B, int *C){ 
	int id = blockIdx.x*blockDim.x + threadIdx.x;  
    // for(int i=0;i<VECTORSIZE;i++){
		int sum = 0;
		for(int j=0; j<VECTORSIZE; j++){
			sum += A[id][j]*B[id];
		}
		C[id] = sum;
	// }    
}

int main(int argc, char **argv){

	int i, j;
	int **A, *B, *C;
	int **a1, *b1, *c1;	
	double exe_time;
	struct timeval stop_time, start_time;
	
	//Allocate and initialize the arrays
	A = (int **)malloc(VECTORSIZE*VECTORSIZE*sizeof(int));
	B = (int *)malloc(VECTORSIZE*sizeof(int));
	C = (int *)malloc(VECTORSIZE*sizeof(int));
	
	//Initialize data to some value
	for(i=0;i<VECTORSIZE;i++){
		for(j=0;j<VECTORSIZE;j++){
			A[i][j] = 1;	
		}
		B[i] = 1;
	}

	gettimeofday(&start_time, NULL);
    hipMalloc(&a1, sizeof(A[0])/sizeof(A));
	hipMemcpy(a1, A, sizeof(A[0])/sizeof(A), hipMemcpyHostToDevice);
	hipMalloc(&b1, sizeof(B[0])/sizeof(B));
	hipMemcpy(b1, B, sizeof(B[0])/sizeof(B), hipMemcpyHostToDevice);
	hipMalloc(&c1, sizeof(C[0])/sizeof(C));
    matrixmul<<< 1,VECTORSIZE >>>(a1, b1, c1);
    hipMemcpy(C, c1, sizeof(C[0])/sizeof(C), hipMemcpyDeviceToHost);
	gettimeofday(&stop_time, NULL);	

	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	//print the data
	printf("\nVector addition output: \n");
	for(i=0;i<VECTORSIZE;i++){
		printf("\t%d", C[i]);	
	}
	printf("\n\n Execution time is = %lf seconds\n", exe_time);
	
	printf("Program exit!\n");
	
	//Free arrays
	free(A); 
	free(B);
	free(C);
}
