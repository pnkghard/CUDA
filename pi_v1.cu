
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 9999999

double pi_cal(){
    double dx, x, area, y, pi;
    dx = 1.0/N;
    for(int i=0; i<N; i++){
        x = i*dx;
        y = sqrt(1-x*x);
        area += y*dx;
    }
    return 4.0*area;
}

int main(){
    struct timeval start, end;
    double pi = pi_cal();
    printf("Value of PI = %lf\n", pi);
    return 0;
}
