
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define N 10000000000


__device__ int isprime(int num){
    for(int i=2; i<=num/2; i++){
        if(num%i==0){
            return 0;
        }
    }
    return 1;
}

__global__ void countprime(int* prime){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=2 && i<=N){
        prime[i] = isprime(i);
    }
}

int main(){
    int size = N*sizeof(int);
	int prime[N], *p, count;
    //initializing array
    for(int i=0; i<N; i++){
        prime[i] = 0;       
    }
    // mem allocation and grid define
    hipMalloc(&p, size);
    int maxThread = 256;
    int blocks = (N/maxThread) + 1;
    //kernal
	countprime<<<N,blocks>>>(p);
    hipMemcpy(prime, p, size, hipMemcpyDeviceToHost);//device to host

    for(int i=0; i<N; i++){
        count += prime[i];
    }

    printf("Total Prime Number : %d\n", count);
}