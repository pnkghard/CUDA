
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define N 999999
#define maxThread 256

__global__ void pi_cal(double* area){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    double x, y, dx=1.0/N;
    if(i<N){
        x = i*dx;
        y = sqrt(1-x*x);
        area[i] = y*dx;
    }
}

int main(){
    double *pi, *p, sum;
    pi = (double *)malloc(N*sizeof(double));
    hipMalloc(&p, N*sizeof(double));
    int blocks = (N/maxThread) + 1;
	pi_cal<<<blocks, maxThread>>>(p);
    hipMemcpy(pi, p, N*sizeof(double), hipMemcpyDeviceToHost);
    for(int i=0; i<N; i++){
        sum+=pi[i];
    }
    sum = 4.0*sum;
    printf("Value of PI = %lf\n", sum);
    return 0;
}