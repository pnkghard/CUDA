
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__device__ int calc_square(int val){
	int val_square;
	val_square = val*val;
	return val_square;
}


__global__ void square(int *array, int* square){
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	square[id] = calc_square(array[id]);
}

int main()
{
	int size = 400 * sizeof(int);
	int a[400], aa[400], *a1, *s;
	int i=0;
	

	//Initialize the vectors
	for(i=0; i<400; i++ )
	{
		a[i] = i;
		aa[i] = 0;
	}


	hipMalloc(&a1, size);
	hipMemcpy(a1, a, size, hipMemcpyHostToDevice);
	hipMalloc(&s, size);
	

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(400, 1);

	square<<< DimGrid, DimBlock >>>(a1, s);
	hipMemcpy(aa, s, size, hipMemcpyDeviceToHost);

	for(i=0; i<400; i++ ){
		printf("\t%d",aa[i]);
	}	
}
