
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 1000000
#define NUM_THDS 256 
/*
                N  PRIME_NUMBER

                1           0
               10           4
              100          25
            1,000         168
           10,000       1,229
          100,000       9,592
        1,000,000      78,498
       10,000,000     664,579
      100,000,000   5,761,455
    1,000,000,000  50,847,534

*/

__global__ void prime_count(int *countd)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int flag = 0;
	int j;
	int count = 0;
	__shared__ int tmp_count[NUM_THDS];
	if(i<N)
        {
        	tmp_count[threadIdx.x] = 0;
        }
	if((i>2)&&(i<N))
	{
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		tmp_count[threadIdx.x] = 1;
        	}
        	else
        	{
            		tmp_count[threadIdx.x] = 0;
        	}
        }
        __syncthreads();
        if(i<N)
        {
        	if(threadIdx.x == 0)
        	{
        		for(j=0;j<NUM_THDS;j++)
        		{
        			if(tmp_count[j] == 1)
        			{
        				count++;
        			}
        		}
        		countd[blockIdx.x] = count;
        	}
        }
}



int main()
{
	int i, j;
	int count, flag, *cnt, *cntd;
	double exe_time;
	struct timeval stop_time, start_time;
	
	count = 1; // 2 is prime. Our loop starts from 3
	
	gettimeofday(&start_time, NULL);
	
	/*for(i=3;i<N;i++)
	{
	 	flag = 0;
		for(j=2;j<i;j++)	
	    	{
		    if((i%j) == 0)
		    {
			    flag = 1;
			    break;
		    }
	    	}
        	if(flag == 0)
        	{
            		count++;
        	}
	}*/
	
	int num_threads_per_block = NUM_THDS;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1; 
	
	cnt = (int *)malloc(num_blocks*sizeof(int));
	hipMalloc(&cntd,num_blocks*sizeof(int));
	
	 
	
	prime_count<<<num_blocks,num_threads_per_block>>>(cntd);
	
	hipMemcpy(cnt,cntd,num_blocks*sizeof(int),hipMemcpyDeviceToHost);
	
	for(i=0;i<num_blocks;i++)
	{
		count+= cnt[i];
	}
	
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	printf("\n Number of prime numbers = %d \n Execution time is = %lf seconds\n", count, exe_time);
	
	free(cnt);
	hipFree(cntd);
	
}

