
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 99999999
#define TB 128

__global__ void calc_area(double dx, double *aread){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double x,y;
	double tmp;
	__shared__ double tmp_area[TB];
	tmp_area[threadIdx.x] = 0.0;
	if(i<N){	
		x = i*dx;
		y = sqrt(1-x*x);
		tmp_area[threadIdx.x] = y*dx;
	}
	__syncthreads();
	if(i<N){
		if(threadIdx.x == 0){
			tmp = 0.0;
			for(int j=0;j<TB;j++){
				tmp += tmp_area[j];
			}
			aread[blockIdx.x] = tmp;
		}
	}
}

int main(){
	int i;
	double total_area, pi, *area, *aread;
	double dx;
	double exe_time;
	struct timeval stop_time, start_time;
	
	dx = 1.0/N;
	total_area = 0.0;
	
	gettimeofday(&start_time, NULL);
	int maxThread = N;
	int blocks = maxThread / TB + 1;
	
	area = (double *)malloc(blocks*sizeof(double));
	hipMalloc(&aread, blocks*sizeof(double));
	
	calc_area<<<blocks,TB>>>(dx, aread);
	
	hipMemcpy(area,aread,blocks*sizeof(double),hipMemcpyDeviceToHost);
	
	for(i=0;i<blocks;i++){
		total_area += area[i];	
	}
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*total_area;
	printf("\n Value of pi is = %lf\n Execution time is = %lf seconds\n", pi, exe_time);
	
	free(area);
	hipFree(aread);
	
}