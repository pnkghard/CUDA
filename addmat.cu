
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>


__global__ void addmatrix(int* md, int* nd, int* pd){
    int myid = threadIdx.y * blockDim.x + threadIdx.x;
    pd[myid] = md[myid] + nd[myid];
}


int main(){
    int size = 20 * 20 * sizeof(int);
    int m[20][20], n[20][20], p[20][20], *md, *nd, *pd;
    int i=0, j=0;
    // initialization of matrix
    for(i=0; i<20; i++){
        for(j=0; j<20; j++){
            m[i][j] = i;
            n[i][j] = i;
            p[i][j] = 0;
        }
    }
    //memory allocation
    hipMalloc(&md, size);
    hipMemcpy(md, m, size, hipMemcpyHostToDevice);//cpu to gpu copy
    hipMalloc(&nd, size);
    hipMemcpy(nd, n, size, hipMemcpyHostToDevice);
    hipMalloc(&pd, size);
    //thread creation
    dim3 dimgrid(1, 1);
    dim3 dimblock(20, 20);
    addmatrix<<<dimgrid, dimblock>>>(md, nd, pd);
    hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);//gpu to cpu copy
    for(i=0; i<20; i++){
        for(j=0; j<20; j++){
            printf("%d\t", p[i][j]);
        }
        printf("\n");
    }
    return 0;
}