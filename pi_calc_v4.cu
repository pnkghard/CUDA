
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

#define N 99999999
#define NUM_THDS 256

__global__ void calc_area(double dx, double *aread)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double x,y;
	if(i<N)
	{	
		x = i*dx;
		y = sqrt(1-x*x);
		aread[i] = y*dx;
	}
}
__global__ void calc_reduction(double *aread, double *aread2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	double tmp;
	if(i<N)
	{
		if(threadIdx.x == 0)
		{
			tmp = 0.0;
			for(int j=0;j<NUM_THDS;j++)
			{
				tmp += aread[blockIdx.x*blockDim.x+j];
			}
			aread2[blockIdx.x] = tmp;
		}
	}

}
int main()
{
	int i;
	double total_area, pi, *area, *aread,*aread2;
	double dx;
	double exe_time;
	struct timeval stop_time, start_time;
	
	dx = 1.0/N;
	total_area = 0.0;
	
	gettimeofday(&start_time, NULL);
	
	int num_threads_per_block = NUM_THDS;
	int total_threads = N;
	int num_blocks = total_threads / num_threads_per_block + 1;
	
	area = (double *)malloc(num_blocks*sizeof(double));
	hipMalloc(&aread, N*sizeof(double));
	hipMalloc(&aread2, num_blocks*sizeof(double));
	
	calc_area<<<num_blocks,num_threads_per_block>>>(dx, aread);
	hipDeviceSynchronize();
	calc_reduction<<<num_blocks,num_threads_per_block>>>(aread,aread2);

	
	hipMemcpy(area,aread2,num_blocks*sizeof(double),hipMemcpyDeviceToHost);
	
	for(i=0;i<num_blocks;i++)
	{
		total_area += area[i];	
	}
	/*for(i=0;i<N;i++)
	{
		x = i*dx;
		y = sqrt(1-x*x);
		area += y*dx;
	}*/
	
	gettimeofday(&stop_time, NULL);	
	exe_time = (stop_time.tv_sec+(stop_time.tv_usec/1000000.0)) - (start_time.tv_sec+(start_time.tv_usec/1000000.0));
	
	pi = 4.0*total_area;
	printf("\n Value of pi is = %.16lf\n Execution time is = %lf seconds\n", pi, exe_time);
	
	free(area);
	hipFree(aread);
	
}

