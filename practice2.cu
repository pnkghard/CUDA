
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void function(double* md, double* nd, double* pd, double al){

	int myid = blockIdx.x*blockDim.x + threadIdx.x;

	pd[myid] = md[myid] + al*nd[myid];
}


int main()
{
	double size = 400 * sizeof(double);
	double a[400], b[400], c[400], alpha, *md, *nd, *pd;
	int i=0;
	
	alpha = 0.001;

	for(i=0; i<400; i++ ){
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMalloc(&md, size);
	hipMemcpy(md, a, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, b, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(400, 1);   


	function<<< DimGrid,DimBlock >>>(md,nd,pd,alpha);

	hipMemcpy(c, pd, size, hipMemcpyDeviceToHost);

	for(i=0; i<400; i++ ){
		printf("\t%lf",c[i]);
	}	

	hipFree(md); 
	hipFree(nd);
	hipFree(pd);
}
